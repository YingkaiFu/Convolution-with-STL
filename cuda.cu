#include <host_defines.h>
#include <hip/hip_runtime.h>
#include "iostream"
using namespace std;
__global__ void add(int a, int b, int *c)//kernel函数，在gpu上运行。
{
    *c = a + b;
}

int cuda() {
    int c;
    int *dev_c;
    hipMalloc((void **) &dev_c, sizeof(int));
    add <<< 1, 1 >>> (2, 7, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    cout << "2 + 7 = " << c << endl;
    hipFree(dev_c);
    return 0;
}